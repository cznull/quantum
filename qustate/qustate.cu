#include "hip/hip_runtime.h"
#include <assert.h>
#include "hip/hip_runtime.h"
#include ""
#include "hipsolver.h"
#include "hipfft/hipfft.h"
#include <gl/glew.h>
#include <cuda_gl_interop.h>
#include "qustate.h"


int m = 0;
int lda = 0;
int lwork = 0;
current2 *d_A = NULL;
current *d_W = NULL;
float *d_ldos = NULL;
current2 *d_work = NULL;
int *devInfo = NULL;
float3 *d_line = NULL;
int *d_ct = NULL;
int *d_img = NULL;
struct hipGraphicsResource *cu_vbo;
struct hipGraphicsResource *cu_pbo;
hipError_t cudaStatus;

hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;
hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

hipsolverHandle_t cusolverH = NULL;
hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
hipfftHandle fftPlan;
hipfftResult fresu;

__global__ void gldos(current2 *vec, current *val, float *img, int size, float s, int el, float es, float eb,float ce) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int i, j;
	float jf;
	for (i = 0; i < size; i++) {
		jf = (val[i] - eb) * s * size*es;
		j = jf;
		if (j < size + el + 1) {
			if (j > -el) {
				float a = vec[i*size + x].x*vec[i*size + x].x + vec[i*size + x].y*vec[i*size + x].y;
				a = a * s;
				for (int k = j - el; k <= j + el; k++) {
					if (k<size && k>-1) {
						img[k*size + x] += exp((k - jf)*(k - jf)*ce)*a;
					}
				}
			}
		}
		else {
			break;
		}
	}
}

__global__ void imgp(float *img, int size, int xl, float cx) {
	int x = threadIdx.x;
	int y = blockIdx.x;
	int i, j;
	float a;
	__shared__ float as[128 + 64 * 3];

	if (x < 64) {
		as[x] = img[y*size + size - 64 + x];
		as[128 + 64 * 2 + x] = img[y*size + x];
		as[128 + 64 + x] = img[y*size + 128 + x];
	}
	__syncthreads();
	as[64 + x] = img[y*size + x];
	__syncthreads();
	a = 0;
	for (j = -xl; j <= xl; j++) {
		a += as[x + 64 + j] * exp(j*j*cx);
	}
	img[y*size + x] = a;

	for (i = 1; i < size / 128 - 1; i++) {
		if (x < 64) {
			as[x] = as[128 + x];
			as[128 + 64 + x] = img[y*size + i * 128 + 128 + x];
		}
		__syncthreads();
		as[64 + x] = img[y*size + i * 128 + x];
		__syncthreads();
		a = 0;
		for (j = -xl; j <= xl; j++) {
			a += as[x + 64 + j] * exp(j*j*cx);
		}
		img[y*size + i * 128 + x] = a;;
	}
	if (x < 64) {
		as[x] = as[128 + x];
		as[128 + 64 + x] = as[128 + 64 * 2 + x];
	}
	__syncthreads();
	as[64 + x] = img[y*size + i * 128 + x];
	__syncthreads();
	a = 0;
	for (j = -xl; j <= xl; j++) {
		a += as[x + 64 + j] * exp(j*j*cx);
	}
	img[y*size + i * 128 + x] = a;
}

__global__ void gline(current2 *vec, current *val, float3 *line, int size, float l, float dx, float m, float es, float eb, float height) {
	int j = threadIdx.x + blockIdx.x * blockDim.x;
	int i = threadIdx.y + blockIdx.y * blockDim.y;
	if (j < size - 1) {
		float y = (val[i] - eb)*es;
		float2 a = { vec[i*size + j].x,vec[i*size + j].y };
		float2 b = { vec[i*size + j + 1].x,vec[i*size + j + 1].y };
		line[i*(size - 1) * 2 + j * 2 + 0] = { l + dx * (j + 0.5f),(a.x*a.x + a.y*a.y)*height *m,-y };
		line[i*(size - 1) * 2 + j * 2 + 1] = { l + dx * (j + 1.5f),(b.x*b.x + b.y*b.y)*height *m,-y };
	}
}

__global__ void gimg(float *ldos, int *ct, int *img, float br) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = log(ldos[i] * br + 1.0f) * 2048.0f + 1024.0f;
	if (j > 4095) {
		j = 4095;
	}
	if (j < 0) {
		j = 0;
	}
	img[i] = ct[j];
}

int cui(int size ,GLuint vbo,GLuint pbo,int *ct) {
	size_t num_bytes;
	m = size;
	lda = size;
	hipSetDevice(0);
	hipMalloc((void**)&devInfo, sizeof(int));
	hipMalloc((void**)&d_A, sizeof(current2) * lda * m);
	hipMalloc((void**)&d_ldos, sizeof(float) * lda * m);
	hipMalloc((void**)&d_W, sizeof(current) * m);
	hipMalloc((void**)&d_ct, sizeof(int) * 4096);
	hipMemcpy(d_ct, ct, sizeof(int) * 4096, hipMemcpyHostToDevice);

#ifdef T_d
	hipfftPlan1d(&fftPlan, size, HIPFFT_Z2Z, size);//d/f
#else
	hipfftPlan1d(&fftPlan, size, HIPFFT_C2C, size);//d/f
#endif

	hipsolverDnCreate(&cusolverH);

#ifdef T_d
	hipsolverDnZheevd_bufferSize(cusolverH, jobz, uplo, m, d_A, lda, d_W, &lwork);
#else
	hipsolverDnCheevd_bufferSize(cusolverH, jobz, uplo, m, d_A, lda, d_W, &lwork);
#endif

	hipMalloc((void**)&d_work, sizeof(current2)*lwork);

	hipGraphicsGLRegisterBuffer(&cu_vbo, vbo, cudaGraphicsMapFlagsWriteDiscard);
	hipGraphicsMapResources(1, &cu_vbo, 0);
	hipGraphicsResourceGetMappedPointer((void **)&d_line, &num_bytes, cu_vbo);
	hipGraphicsUnmapResources(1, &cu_vbo, 0);

	hipGraphicsGLRegisterBuffer(&cu_pbo, pbo, cudaGraphicsMapFlagsWriteDiscard);
	hipGraphicsMapResources(1, &cu_pbo, 0);
	hipGraphicsResourceGetMappedPointer((void **)&d_img, &num_bytes, cu_pbo);
	hipGraphicsUnmapResources(1, &cu_pbo, 0);
	return 0;
}

int cueigen(void *hm, void *val, void *vec, int size) {
	hipMemcpy(d_A, hm, sizeof(current2) * lda * m, hipMemcpyHostToDevice);

#ifdef T_d
	hipsolverDnZheevd(cusolverH, jobz, uplo, m, d_A, lda, d_W, d_work, lwork, devInfo);//d/f
	hipfftExecZ2Z(fftPlan, d_A, d_A, HIPFFT_BACKWARD);//d/f
#else
	hipsolverDnCheevd(cusolverH, jobz, uplo, m, d_A, lda, d_W, d_work, lwork, devInfo);//d/f
	hipfftExecC2C(fftPlan, d_A, d_A, HIPFFT_BACKWARD);//d/f
#endif

	hipMemcpy(val, d_W, sizeof(current)*m, hipMemcpyDeviceToHost);
	hipMemcpy(vec, d_A, sizeof(current2)*lda*m, hipMemcpyDeviceToHost);
	return 0;
}

int cuimg(float *img, int size, float s, float el, float xl, float es, float eb, float br) {
	hipError_t cudaStatus;
	hipMemset(d_ldos, 0, sizeof(float)*size*size);
	gldos << <size / 128, 128 >> > (d_A, d_W, d_ldos, size, s, el, es, eb, -4.0 / el / el);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		return 1;
	}
	imgp << <size, 128 >> > (d_ldos, size, xl, -4.0 / xl / xl);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		return 1;
	}
	hipGraphicsMapResources(1, &cu_pbo, 0);
	gimg << <size*size / 128, 128 >> > (d_ldos, d_ct, d_img, br / el / xl);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		return 1;
	}
	hipGraphicsUnmapResources(1, &cu_pbo, 0);
	//hipMemcpy(img, d_ldos, sizeof(float)*size*size, hipMemcpyDeviceToHost);
	return 0;
}

int culine(int size, float l, float r, float es, float eb, float height) {
	hipGraphicsMapResources(1, &cu_vbo, 0);
	gline << < dim3(size / 128, size, 1), dim3(128, 1, 1) >> > (d_A, d_W, d_line, size, l, (r - l)*(1.0 / size), 1.0 / (r - l), es, eb, height);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		return 1;
	}
	hipGraphicsUnmapResources(1, &cu_vbo, 0);
	return 0;
}